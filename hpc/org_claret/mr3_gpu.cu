
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#define NTHRE   (64)

#define MALLOC_AND_D2F(n,double_mem,float_mem) \
  (float_mem)=(float *)malloc(sizeof(float)*n);	\
  {int i;for(i=0;i<(n);i++) (float_mem)[i]=(double_mem)[i];}

//__global__
void GPU_kernel(float *x, int n, int *atype, int nat, float *pol, float *sigm, float *ipotro,
		float *pc, float *pd, float *zz, int tblno, float xmax, int periodicflag, 
		float *force)
{
  int tid; // set thread index among grid
  int i;   // set global thread index
  int j,k,t;
  float dn2,r,inr,inr2,inr4,inr8,d3,dr[3],fi[3];
  float pb=(float)(0.338e-19/(14.39*1.60219e-19)),dphir; 

  for(i=0; i<n; i++){
    for(k=0; k<3; k++) fi[k] = 0.0f;
    for(j=0; j<n; j++){
      dn2 = 0.0f;
      for(k=0; k<3; k++){
	dr[k] = x[i*3+k] - x[j*3+k];
	dn2  += dr[k] * dr[k];
      }
      if(dn2 != 0.0f){
	r     = sqrtf(dn2);
	inr   = 1.0f  / r;
	inr2  = inr  * inr;
	inr4  = inr2 * inr2;
	inr8  = inr4 * inr4;
	t     = atype[i] * nat + atype[j];
	d3    = pb * pol[t] * expf( (sigm[t] - r) * ipotro[t]);
	dphir = ( d3 * ipotro[t] * inr
		  - 6.0f * pc[t] * inr8
		  - 8.0f * pd[t] * inr8 * inr2
		  + inr2 * inr * zz[t] );
	for(k=0; k<3; k++) fi[k] += dphir * dr[k];
      }
    }
    for(k=0; k<3; k++) force[i*3+k] = fi[k];
  }
}

extern "C"
void MR3calcnacl_GPU(double x[], int n, int atype[], int nat,
		     double pol[], double sigm[], double ipotro[],
		     double pc[], double pd[], double zz[],
		     int tblno, double xmax, int periodicflag,
		     double force[])
{
  int i,*d_atype;
  float *f_x,*f_pol,*f_sigm,*f_ipotro,*f_pc,*f_pd,*f_zz,*f_force,xmaxf=xmax;
  float *d_x,*d_pol,*d_sigm,*d_ipotro,*d_pc,*d_pd,*d_zz,*d_force;

  if(periodicflag!=0){
    fprintf(stderr,"** error : periodicflag = %d is not supported **\n",periodicflag);
    exit(1);
  }

  MALLOC_AND_D2F(n*3,x,f_x);
  MALLOC_AND_D2F(nat*nat,pol,f_pol);
  MALLOC_AND_D2F(nat*nat,sigm,f_sigm);
  MALLOC_AND_D2F(nat*nat,ipotro,f_ipotro);
  MALLOC_AND_D2F(nat*nat,pc,f_pc);
  MALLOC_AND_D2F(nat*nat,pd,f_pd);
  MALLOC_AND_D2F(nat*nat,zz,f_zz);
  f_force=(float *)malloc(sizeof(float)*n*3);

  // allocate global memory: d_x, d_pol, d_sigm, d_ipotro, d_pc, d_pd, d_zz, d_force, d_atype

  
  // copy from host to GPU: d_x, d_pol, d_sigm, d_ipotro, d_pc, d_pd, d_zz, d_atype


  // call GPU kernel
  dim3 threads(NTHRE);
  dim3 grid((n+NTHRE-1)/NTHRE);
  GPU_kernel(f_x,n,atype,nat,f_pol,f_sigm,f_ipotro,
	     f_pc,f_pd,f_zz,tblno,xmaxf,periodicflag,f_force);

  // copy GPU result to host

  
  for(i=0;i<n*3;i++) force[i]=f_force[i];

  free(f_x);
  free(f_pol);
  free(f_sigm);
  free(f_ipotro);
  free(f_pc);
  free(f_pd);
  free(f_zz);
  free(f_force);

  // free allocated global memory

}
