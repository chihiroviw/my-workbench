
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>


#define NMAX    8192
#define NTHRE   (64)
#define ATYPE        8
#define ATYPE2    (ATYPE * ATYPE)

typedef struct {
    float r[3];
    int atype;
} VG_XVEC;

typedef struct {
    float pol;
    float sigm;
    float ipotro;
    float pc;
    float pd;
    float zz;
} VG_MATRIX;

__constant__ VG_MATRIX d_matrix[sizeof(VG_MATRIX)*2*2];


__device__ __inline__ 
void inter(float xj[3], float xi[3], float fi[3], 
	        //int t, float xmax, float xmax1, VG_MATRIX* d_matrix){
	        int t, float xmax, float xmax1){

    int k;
    float dn2,r,inr,inr2,inr4,inr8,d3,dr[3];
    float pb=(float)(0.338e-19/(14.39*1.60219e-19)),dphir;

    dn2 = 0.0f;
    for(k=0; k<3; k++){
        dr[k]  = xi[k] - xj[k];
        dr[k] -= rintf(dr[k] * xmax1) * xmax;
        dn2   += dr[k] * dr[k];
    }

    //if(dn2 != 0.0f){
        r     = sqrtf(dn2);
        inr   = 1.0f / r;
        inr2  = inr  * inr;
        inr4  = inr2 * inr2;
        inr8  = inr4 * inr4;
        d3    = pb * d_matrix[t].pol * expf( (d_matrix[t].sigm - r) * d_matrix[t].ipotro);
        dphir = ( d3 * d_matrix[t].ipotro * inr
	            - 6.0f * d_matrix[t].pc * inr8
	            - 8.0f * d_matrix[t].pd * inr8 * inr2
	            + inr2 * inr * d_matrix[t].zz );
        //for(k=0; k<3; k++) fi[k] += dphir * dr[k];
    //}
    if(dn2 == 0.0f) dphir = 0; 
    for(k=0; k<3; k++) fi[k] += dphir * dr[k];
}

extern "C" __global__
void nacl_kernel_gpu(VG_XVEC *x, int n, int nat, float xmax, float *fvec){
//void nacl_kernel_gpu(VG_XVEC *x, int n, int nat, float xmax, float *fvec,VG_MATRIX* d_matrix){
//void nacl_kernel_gpu(float* r,int* atype, int n, int nat, float xmax, float *fvec,VG_MATRIX* d_matrix){
    int tid = threadIdx.x;
    int i = blockIdx.x * NTHRE + tid;
    int j,k;
    float fi[3],xmax1=1.0f/xmax;
    int atypei;
    float xi[3];

    
    for(k=0; k<3; k++) fi[k] = 0.0f;

    //for(int tt=0; tt<11; tt++){

    for(k=0; k<3; k++) xi[k] = x[i].r[k];
    atypei = x[i].atype * nat;

    //for(k=0; k<3; k++) xi[k] = r[i*3+k];
    //atypei = atype[i] * nat;
    //for (j = 0; j < n; j++){
    //    inter(x[j].r, xi, fi, atypei + x[j].atype, xmax, xmax1, d_matrix);
        //inter(&r[j*3], xi, fi, atypei + atype[j], xmax, xmax1, d_matrix);
    //}
    
    
    //sheared
    __shared__ VG_XVEC s_xj[NTHRE];
    for (j = 0; j < n; j+=64){

        //copy to shared memory
        __syncthreads();
        s_xj[threadIdx.x] = x[j+threadIdx.x]; 
        __syncthreads();

        for(int js=0; js<min(64,n-j); js++){
            //inter(s_xj[js].r, xi, fi, atypei + s_xj[js].atype, xmax, xmax1,d_matrix);
            inter(s_xj[js].r, xi, fi, atypei + s_xj[js].atype, xmax, xmax1);
        }
    }

    //}

    if(i<n) for(k=0; k<3; k++) fvec[i*3+k] = fi[k];;
}

extern "C"
void MR3calcnacl_GPU(double x[], int n, int atype[], int nat,
		     double pol[], double sigm[], double ipotro[],
		     double pc[], double pd[], double zz[],
		     int tblno, double xmax, int periodicflag,
		     double force[])
{

    int i,j;
    static VG_XVEC *d_x=NULL;
    static float *d_force=NULL; 


    //VG_MATRIX *d_matrix;
    //float *d_r;
    //int *d_atype;

    float xmaxf;
    VG_MATRIX *matrix=(VG_MATRIX *)force;
    static VG_XVEC   *vec=NULL;
    if((periodicflag & 1)==0) xmax*=2.0;
    xmaxf=xmax;
    static float *forcef=NULL;
    static int n_bak=0;
        
    // allocate global memory
    
    if(n!=n_bak){
    //if(true){
        // free and allocate global memory
        int nalloc;
        static int nalloc_bak=0;

        if(n>NMAX) nalloc=n;
        else       nalloc=NMAX;

        if(nalloc!=nalloc_bak){
        //if(true){
            (hipFree(d_x));
            (hipFree(d_force));
            (hipFree(d_matrix));
            (hipMalloc((void**)&d_x,sizeof(VG_XVEC)*(nalloc+NTHRE)));
            (hipMalloc((void**)&d_force,sizeof(float)*nalloc*3));
            (hipMalloc((void**)&d_matrix,sizeof(VG_MATRIX)*nat*nat));
            
            //(cudaFree(d_r));
            //(cudaFree(d_atype));
            //(cudaMalloc((void**)&d_r,sizeof(float)*3*nalloc));
            //(cudaMalloc((void**)&d_atype,sizeof(int)*nalloc));

            free(vec);
            vec=(VG_XVEC *)malloc(sizeof(VG_XVEC)*(nalloc+NTHRE));
            free(forcef);
            forcef=(float *)malloc(sizeof(float)*nalloc*3);
            bzero(forcef,sizeof(float)*nalloc*3);
            nalloc_bak=nalloc;
        }
        // set matrix
        for(i=0;i<nat;i++){
            for(j=0;j<nat;j++){
	            matrix[i*nat+j].pol=(float)(pol[i*nat+j]);
	            matrix[i*nat+j].sigm=(float)(sigm[i*nat+j]);
                matrix[i*nat+j].ipotro=(float)(ipotro[i*nat+j]);
                matrix[i*nat+j].pc=(float)(pc[i*nat+j]);
                matrix[i*nat+j].pd=(float)(pd[i*nat+j]);
                matrix[i*nat+j].zz=(float)(zz[i*nat+j]);
            }
        }

        //constant memory
        (hipMemcpyToSymbol(HIP_SYMBOL(d_matrix),matrix,sizeof(VG_MATRIX)*nat*nat));
        //(cudaMemcpy(d_matrix,matrix,sizeof(VG_MATRIX)*nat*nat,cudaMemcpyHostToDevice));

        n_bak=n;
    }
    
    // copy from host to GPU
    for(i=0;i<(n+NTHRE-1)/NTHRE*NTHRE;i++){
        if(i<n){
            for(j=0;j<3;j++){
	            vec[i].r[j]=x[i*3+j];
            }
            vec[i].atype=atype[i];

        }else{
            for(j=0;j<3;j++){
	            vec[i].r[j]=0.0f;
            }
            vec[i].atype=0;
        }
    }
    
    (hipMemcpy(d_x,vec,sizeof(VG_XVEC)*((n+NTHRE-1)/NTHRE*NTHRE),hipMemcpyHostToDevice));
    
    /*
    float *r_vec = (float*)malloc(sizeof(float)*3*n);
    for(int i=0; i<n; i++){
        r_vec[i*3+0] = x[i*3+0];
        r_vec[i*3+1] = x[i*3+1];
        r_vec[i*3+2] = x[i*3+2];
    }
    (cudaMemcpy(d_r,r_vec,sizeof(float)*3*n,cudaMemcpyHostToDevice));
    (cudaMemcpy(d_atype,atype,sizeof(int)*n,cudaMemcpyHostToDevice));
    */
    

    // call GPU kernel
    dim3 threads(NTHRE);
    dim3 grid((n+NTHRE-1)/NTHRE);
    nacl_kernel_gpu<<< grid, threads >>>(d_x,n,nat,xmaxf,d_force);
    //nacl_kernel_gpu<<< grid, threads >>>(d_x,n,nat,xmaxf,d_force,d_matrix);
    //nacl_kernel_gpu<<< grid, threads >>>(d_r,d_atype,n,nat,xmaxf,d_force,d_matrix);

    // copy GPU result to host
    (hipMemcpy(forcef,d_force,sizeof(float)*n*3,hipMemcpyDeviceToHost));
    for(i=0;i<n;i++){ 
        for(j=0;j<3;j++){ 
            force[i*3+j]=forcef[i*3+j];
        }
    }
}
