
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE    (3072)
#define NTHRE   (64)

float in[SIZE];
float out[SIZE];

__global__ void abs_kernel(int n, float *in, float *out)
{
  /* Calculate absolute values */
  int i=blockIdx.x * blockDim.x + threadIdx.x, loop;

  for(loop=0;loop<1000000;loop++){
    if (in[i] > 0) {
      out[i] = in[i];
    } else {
      out[i] = in[i] * -1;
    }
  }
}

int main(int argc, char **argv)
{
  int i;
  float *d_in, *d_out;

  for(i=0;i<SIZE;i+=2){
    in[i]=i+1;
    in[i+1]=-(i+2);
  }

  /* allocate and copy GPU memory */
  hipMalloc((void **)&d_in, sizeof(float)*SIZE);
  hipMalloc((void **)&d_out,sizeof(float)*SIZE);
  hipMemcpy(d_in,in,sizeof(float)*SIZE,hipMemcpyHostToDevice);

  /* call GPU kernel */
  dim3 threads(NTHRE);
  dim3 grid((SIZE + NTHRE - 1)/NTHRE);
  abs_kernel<<<grid, threads>>>(SIZE, d_in, d_out);

  /* copy result to CPU memory */
  hipMemcpy(out,d_out,sizeof(float)*SIZE,hipMemcpyDeviceToHost);

  for (i = SIZE-3; i < SIZE; i++) {
    printf("in [%02d]=%0.0f\n", i, in[i]);
    printf("out[%02d]=%0.0f\n", i, out[i]);
  }
  hipFree(d_in); hipFree(d_out);  
  return 0;
}
