
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#define EPS2    (0.03f*0.03f)
#define NTHRE   (64)

__global__
void float_force_optimized_sub(int n,                // input  : number of particles
			       float posf[][4],      // input  : position and mass of particles
			       float forcef[][4])    // output : force and potential of particles
{
  int tid = threadIdx.x;
  int i = blockIdx.x * NTHRE + tid;
  int j,k;
  float dr[3],r_1,dtmp,r2,fi[4],sqrtfeps2=1.0f/sqrtf(EPS2);
  if(i<n){
	for(k=0;k<4;k++) fi[k]=0.0f;
	for(j=0;j<n;j++){
		r2=EPS2;

		for(k=0;k<3;k++){
			dr[k]=posf[i][k]-posf[j][k];
			r2+=dr[k]*dr[k];
		}

		r_1=1.0f/sqrtf(r2);
	        dtmp=posf[j][3]*r_1;
	        fi[3]+=dtmp;
	        dtmp*=r_1*r_1;

	        for(k=0;k<3;k++) fi[k]-=dtmp*dr[k];
	}

	fi[3]-=posf[i][3]*sqrtfeps2;

	for(k=0;k<4;k++) forcef[i][k]=fi[k]*posf[i][3];
  }
}

/*
__global__
void float_force_optimized_sub(int n,                // input  : number of particles
			       float posf[][4],      // input  : position and mass of particles
			       float forcef[][4])    // output : force and potential of particles
{
  int tid = threadIdx.x;
  int i = blockIdx.x * NTHRE + tid;
  int j,k;
  float dr[3],r_1,dtmp,r2,fi[4],sqrtfeps2=1.0f/sqrtf(EPS2);

  if(i<n){
    for(k=0;k<4;k++) fi[k]=0.0f;
    for(j=0;j<n;j++){
      r2=EPS2;
      for(k=0;k<3;k++){
	dr[k]=posf[i][k]-posf[j][k];
	r2+=dr[k]*dr[k];
      }
      r_1=1.0f/sqrtf(r2);
      dtmp=posf[j][3]*r_1;
      fi[3]+=dtmp;
      dtmp*=r_1*r_1;
      for(k=0;k<3;k++) fi[k]-=dtmp*dr[k];
    }
    fi[3]-=posf[i][3]*sqrtfeps2;
    for(k=0;k<4;k++) forcef[i][k]=fi[k]*posf[i][3];
  }
}
*/


extern "C"
void float_force_optimized(int n,                // input  : number of particles
			   double pos[][4],      // input  : position and mass of particles
			   double force[][4])    // output : force and potential of particles
{
  int i,j,k;
  float (*posf)[4],(*forcef)[4];
  float (*d_posf)[4], (*d_forcef)[4];

  // Allocate and copy positions and masses
  posf=(float (*)[4])malloc(sizeof(float)*n*4);
  forcef=(float (*)[4])malloc(sizeof(float)*n*4);
  hipMalloc((void **)&d_posf,sizeof(float)*n*4);
  hipMalloc((void **)&d_forcef,sizeof(float)*n*4);
  for(j=0;j<n;j++) for(k=0;k<4;k++) posf[j][k]=(float)pos[j][k];
  hipMemcpy(d_posf,posf,sizeof(float)*n*4,hipMemcpyHostToDevice);

  // Calculate force and potential with GPU
  dim3 threads(NTHRE);
  dim3 grid((n+NTHRE-1)/NTHRE);
  float_force_optimized_sub<<<grid, threads>>>(n,d_posf,d_forcef);

  // Copy back to double array
  hipMemcpy(forcef,d_forcef,sizeof(float)*n*4,hipMemcpyDeviceToHost);
  for(i=0;i<n;i++) for(k=0;k<4;k++) force[i][k]=(double)forcef[i][k];

  free(posf);free(forcef);
  hipFree(d_posf);hipFree(d_forcef);
}

